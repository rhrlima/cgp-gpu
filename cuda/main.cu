#include "hip/hip_runtime.h"
#include <stdio.h>

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/functional.h>

#include "cgp.cuh"

int main() {

	unsigned int seed = time(NULL);
	srand(seed);

	char dataset_file[100];

	struct dataset *data;
	struct parameters *params;
	// struct chromosome *chromo, *best;
	struct chromosome *best;

	strcpy(dataset_file, "datasets/symbolic2_1024.data");	//x*x + x+x | (9, 2, 4)
	//strcpy(dataset_file, "datasets/symbolic3.data");	//(x0+x1) + (x0*x1) + (-x0)*(x1*x1) | (8, 2, 4)
	//strcpy(dataset_file, "datasets/symbolic4.data");  //x0*x1*x1 + x2*x1 + x*3

	data = loadDataset(dataset_file);

	params = initialiseParameters(9, 2, 4, data);//numNodes, maxArity, numFunctions

	printf("Dataset: '%s'\n", dataset_file);

	printParameters(params);

	// printf("Running CGP\n");
	// chromo = executeCGP(params, data, 10000);

	// printf("Best solution found\n");
	// printChromosome(chromo);

	int array[] = {1, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 2, 0, 2, 4, 1, 4, 1, 2, 2, 5, 3, 2, 3, 6}; //best for dataset 2
	// // int array[] = {0, 0, 1, 2, 0, 1, 1, 0, 0, 1, 4, 0, 2, 1, 1, 2, 5, 6, 0, 2, 3, 0, 8, 7, 9}; //best for dataset 3
	// // int array[] = {2, , 1, 2};
	best = createChromosomeFromArray(params, array);
	calculateFitness(best, data);

	// printf("Best hardcoded\n");
	printChromosome(best);

	/* CUDA */
	int size = best->numNodes * (best->arity + 1) + best->numOutputs;
	int *solution = (int*)malloc(size * sizeof(int));
	createArrayFromChromosome(*best, solution);
	for(int i=0; i<size; i++)
		printf("%d ", solution[i]);
	printf("\n");
	

	int *d_solution;
	double *d_data_inputs;//, *d_data_outputs;

	printf("Cuda Malloc\n");
	hipMalloc(&d_solution, 28 * sizeof(int));
	hipMalloc(&d_data_inputs, data->numSamples * sizeof(double));
	// hipMalloc(&d_data_outputs, data->numSamples * sizeof(double));

	printf("Memory Copy\n");
	hipMemcpy(d_solution, solution, 28 * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_data_inputs, data->inputs, data->numSamples * sizeof(double), hipMemcpyHostToDevice);
	// // hipMemcpy(d_data_outputs, data->outputs, data->numSamples * sizeof(double), hipMemcpyHostToDevice);

	// //setUpChromosomeData<<<4, 256>>>(dInPtr, dOutPrt, d_data_inputs, d_data_outputs, data->numSamples);

	thrust::device_vector<double> outputs(data->numSamples);
	double *dOutPrt = thrust::raw_pointer_cast(outputs.data());
	printf("A\n");
	teste<<<4, 256>>>(d_solution, d_data_inputs, dOutPrt, data->numSamples, 1, best->numNodes);
	printf("B\n");
	double error = 0.0;
	for(int i=0; i<data->numSamples; i++) {
		error += fabs(outputs[i] - data->outputs[i]);
	}
	printf("Fitness: %f\n", error);

	// freeChromosome(chromo);
	freeChromosome(best);

	freeDataset(data);
	free(params);

	return 0;
}