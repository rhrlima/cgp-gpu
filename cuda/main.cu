#include "hip/hip_runtime.h"
#include <stdio.h>

#include <thrust/reduce.h>
#include <thrust/execution_policy.h>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include "cgp.cuh"

int main() {

	unsigned int seed = time(NULL);
	srand(seed);

	char dataset_file[100];

	struct dataset *data;
	struct parameters *params;
	// struct chromosome *chromo, *best;
	struct chromosome *best;

	strcpy(dataset_file, "datasets/symbolic2_1024.data");	//x*x + x+x | (9, 2, 4)
	//strcpy(dataset_file, "datasets/symbolic3.data");	//(x0+x1) + (x0*x1) + (-x0)*(x1*x1) | (8, 2, 4)
	//strcpy(dataset_file, "datasets/symbolic4.data");  //x0*x1*x1 + x2*x1 + x*3

	data = loadDataset(dataset_file);

	params = initialiseParameters(9, 2, 4, data);//numNodes, maxArity, numFunctions

	// printf("Dataset: '%s'\n", dataset_file);

	// printParameters(params);

	// printf("Running CGP\n");
	// chromo = executeCGP(params, data, 10000);

	// printf("Best solution found\n");
	// printChromosome(chromo);

	int array[] = {1, 0, 0, 2, 0, 0, 0, 0, 0, 0, 0, 0, 1, 0, 2, 0, 2, 4, 1, 4, 1, 2, 2, 5, 3, 2, 3, 6}; //best for dataset 2
	// // int array[] = {0, 0, 1, 2, 0, 1, 1, 0, 0, 1, 4, 0, 2, 1, 1, 2, 5, 6, 0, 2, 3, 0, 8, 7, 9}; //best for dataset 3
	// // int array[] = {2, , 1, 2};
	best = createChromosomeFromArray(params, array);
	calculateFitness(best, data);

	// printf("Best hardcoded\n");
	printChromosome(best);

	/* test */
	best->fitness = -1;

	thrust::device_vector<double> outputs(data->numSamples);
	double *out = thrust::raw_pointer_cast(outputs.data());

	//4 x 256 = 1024 samples
	// thrust::fill(fitnesses.begin(), fitnesses.end(), 0);

	int numThreads = 256;
	int numBlocks = ceil((float)data->numSamples/numThreads);
	cudaCalculateFitnesses<<<4, 256>>>(*best, out, data->inputs, data->numSamples);
	
	for(int i=0; i<data->numSamples; i++) {
		printf("%4d %6.2f\n", i, (double) outputs[i]);
	}

	// double error = thrust::reduce(fitnesses.begin(), fitnesses.end());
	// printf("Fitness: %f\n", error);

	// freeChromosome(chromo);
	freeChromosome(best);

	freeDataset(data);
	free(params);

	//---------------------

	return 0;
}