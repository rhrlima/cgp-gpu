#include "hip/hip_runtime.h"
#ifndef HEADER_CGP_
#define HEADER_CGP_

#include "cgp.cuh"

#define NUMNODES 100
#define NTHREADS 1024

#define ADD 0
#define SUB 1
#define MUL 2
#define DIV 3


struct node *createNode(int numInputs, int numNodes, int arity, int numFunctions, int nodePosition) {

	struct node *n;
	int i;

	/* allocate memory for node */
	n = (struct node*)malloc(sizeof(struct node));

	/* allocate memory for the node's inputs and connection weights */
	n->inputs = (int*)malloc(arity * sizeof(int));

	/* set the node's function */
	n->function = randint(0, numFunctions);

	/* set as active by default */
	n->active = 1;

	/* set the nodes inputs and connection weights */
	for (i = 0; i < arity; i++) {
		n->inputs[i] = getRandomNodeInput(numInputs, nodePosition);
	}

	/* set the output of the node to zero*/
	n->output = 0;

	/* set the arity of the node */
	n->maxArity = arity;
	n->actArity = arity;

	return n;
}


void copyNode(struct node *dst, struct node *src) {

	int i;

	dst->function = src->function;

	for(i = 0; i < src->maxArity; i++) {
		dst->inputs[i] = src->inputs[i];
	}

	dst->active = src->active;
	dst->output = src->output;
	dst->maxArity = src->maxArity;
	dst->actArity = src->actArity;
}


void freeNode(struct node *node) {

	if (node == NULL) {
		printf("Warning: freeing NULL node avoided\n");
		return;
	}

	free(node->inputs);
	free(node);
}


struct chromosome *createChromosome(struct parameters *params) {

	struct chromosome *chromo;
	int i;

	chromo = (struct chromosome*)malloc(sizeof(struct chromosome));
	chromo->nodes = (struct node**)malloc(params->numNodes * sizeof(struct node*));
	chromo->outputNodes = (int*)malloc(params->numOutputs * sizeof(int));
	chromo->activeNodes = (int*)malloc(params->numNodes * sizeof(int));
	chromo->outputValues = (double*)malloc(params->numOutputs * sizeof(double));
	for (i = 0; i < params->numNodes; i++) {
		chromo->nodes[i] = createNode(params->numInputs, params->numNodes, params->arity, params->numFunctions, i);
	}
	for (i = 0; i < params->numOutputs; i++) {
		chromo->outputNodes[i] = getRandomChromosomeOutput(params->numInputs, params->numNodes);
	}

	chromo->numInputs = params->numInputs;
	chromo->numNodes = params->numNodes;
	chromo->numOutputs = params->numOutputs;
	chromo->arity = params->arity;

	/* set the number of active node to the number of nodes (all active) */
	chromo->numActiveNodes = params->numNodes;

	/* set the fitness to initial value */
	chromo->fitness = -1;

	/* set the active nodes in the newly generated chromosome */
	setChromosomeActiveNodes(chromo);

	/* used interally when executing chromosome */
	chromo->nodeInputsHold = (double*)malloc(params->arity * sizeof(double));

	return chromo;
}


struct chromosome *createChromosomeFromArray(struct parameters *params, int *array) {

	struct chromosome *chromo;
	int i, j;

	chromo = (struct chromosome*)malloc(sizeof(struct chromosome));
	chromo->nodes = (struct node**)malloc(params->numNodes * sizeof(struct node*));
	chromo->outputNodes = (int*)malloc(params->numOutputs * sizeof(int));
	chromo->activeNodes = (int*)malloc(params->numNodes * sizeof(int));
	chromo->outputValues = (double*)malloc(params->numOutputs * sizeof(double));

	for (i = 0; i < params->numNodes; i++) {

		chromo->nodes[i] = (struct node*)malloc(sizeof(struct node));
		chromo->nodes[i]->inputs = (int*)malloc(params->arity * sizeof(int));
		chromo->nodes[i]->function = array[i * (params->arity + 1)];

		for (j = 0; j < params->arity; j++) {
			chromo->nodes[i]->inputs[j] = array[i * (params->arity + 1) + j + 1];
		}

		chromo->nodes[i]->active = 1;
		chromo->nodes[i]->output = 0;

		chromo->nodes[i]->maxArity = params->arity;
		chromo->nodes[i]->actArity = params->arity;
	}

	for (i = 0; i < params->numOutputs; i++) {
		chromo->outputNodes[i] = array[params->numNodes * (params->arity + 1) + i];
	}

	chromo->numInputs = params->numInputs;
	chromo->numNodes = params->numNodes;
	chromo->numOutputs = params->numOutputs;
	chromo->arity = params->arity;

	chromo->numActiveNodes = params->numNodes;
	chromo->fitness = -1;

	setChromosomeActiveNodes(chromo);

	chromo->nodeInputsHold = (double*)malloc(params->arity * sizeof(double));

	return chromo;
}


void copyChromosome(struct chromosome *dst, struct chromosome *src) {

	int i;

	dst->numInputs = src->numInputs;
	dst->numOutputs = src->numOutputs;
	dst->numNodes = src->numNodes;
	dst->numActiveNodes = src->numActiveNodes;
	dst->arity = src->arity;
	dst->fitness = src->fitness;

	for(i = 0; i < src->numNodes; i++) {
		copyNode(dst->nodes[i], src->nodes[i]);
		dst->activeNodes[i] = src->activeNodes[i];
	}

	for(i = 0; i < src->numOutputs; i++) {
		dst->outputNodes[i] = src->outputNodes[i];
		dst->outputValues[i] = src->outputValues[i];
	}
}


void freeChromosome(struct chromosome *chromo) {

	if (chromo == NULL) {
		printf("Warning: Avoiding free NULL chromosome.\n");
		return;
	}

	int i;

	for(i = 0; i < chromo->numNodes; i++) {
		freeNode(chromo->nodes[i]);
	}

	free(chromo->nodes);
	free(chromo->outputNodes);
	free(chromo->activeNodes);
	free(chromo->outputValues);
	free(chromo->nodeInputsHold);
	free(chromo);
}


int getRandomNodeInput(int numChromoInputs, int nodePosition) {
	/* pick any previous node including inputs */
	return randint(0, numChromoInputs + nodePosition);
}


int getRandomChromosomeOutput(int numInputs, int numNodes) {

	/* returns any previous node */
	return randint(0, numInputs + numNodes);
}


void setChromosomeActiveNodes(struct chromosome *chromo) {

	int i, j;

	chromo->numActiveNodes = 0;

	for(i = 0; i < chromo->numNodes; i++) {
		chromo->activeNodes[i] = 0;
	}

	for(i = 0; i < chromo->numOutputs; i++) {
		if (chromo->outputNodes[i] >= chromo->numInputs)
			chromo -> activeNodes[ chromo->outputNodes[i] - chromo->numInputs ] = 1;
	}

	for(i = chromo->numNodes-1; i >= 0; i--) {
		if (chromo->activeNodes[i]) {
			for (j=0; j<chromo->arity; j++) {
				if (chromo->nodes[i]->inputs[j] >= chromo->numInputs)
					chromo->activeNodes[ chromo->nodes[i]->inputs[j] - chromo->numInputs ] = 1;
			}
			chromo->numActiveNodes++;
		}
	}
}


void printChromosome(struct chromosome *chromo) {
	int i, j;

	if (chromo == NULL) {
		printf("Warning: Avoiding print NULL chromosome.\n");
		return;
	}

	for(i = 0; i < chromo->numNodes; i++) {

		printf("%d ", chromo->nodes[i]->function);

		for(j = 0; j < chromo->arity; j++) {

			printf("%d ", chromo->nodes[i]->inputs[j]);
		}
	}
	printf("| ");
	for(i = 0; i < chromo->numOutputs; i++) {

		printf("%d ", chromo->outputNodes[i]);
	}
	printf("| ");
	for(i = 0; i < chromo->numNodes; i++){

		printf("%d ", chromo->activeNodes[i]);
	}
	printf("| %.2f\n", chromo->fitness);
}



/* -------------------------------------------------- */


void executeChromosome(struct chromosome *chromo, double inputs) {

	int i, j;
	int nodeInputLocation;
	int currentActiveNode;
	int currentActiveNodeFuction;
	int nodeArity;

	int numInputs = chromo->numInputs;
	int numNodes = chromo->numNodes;
	int numOutputs = chromo->numOutputs;

	/* error checking */
	if (chromo == NULL) {
		printf("Error: cannot execute uninitialised chromosome.\n Terminating CGP-Library.\n");
		exit(0);
	}

	/* for all of the active nodes */
	for (i = 0; i < numNodes; i++) {

		if (chromo->activeNodes[i]) {

			currentActiveNode = i;
			nodeArity = chromo->nodes[currentActiveNode]->actArity;

			for(j = 0; j < nodeArity; j++) {
				
				nodeInputLocation = chromo->nodes[currentActiveNode]->inputs[j];

				/* verify if the input location is a node or real input */
				if(nodeInputLocation < numInputs) {
					chromo->nodeInputsHold[j] = inputs;
				}
				else {
					chromo->nodeInputsHold[j] = chromo->nodes[nodeInputLocation - numInputs]->output;
				}
			}

			/* get the functionality of the active node under evaluation */
			currentActiveNodeFuction = chromo->nodes[currentActiveNode]->function;

			/* calculate the output of the active node under evaluation */
			//melhorar depois
			double output = 0.0;
			switch(currentActiveNodeFuction) {
				case ADD:
					output = chromo->nodeInputsHold[0] + chromo->nodeInputsHold[1];
					break;
				case SUB:
					output = chromo->nodeInputsHold[0] - chromo->nodeInputsHold[1];
					break;
				case MUL:
					output = chromo->nodeInputsHold[0] * chromo->nodeInputsHold[1];
					break;
				case DIV:
					output = chromo->nodeInputsHold[0] / chromo->nodeInputsHold[1];
					break;
			}

			if (isnan(output) != 0) output = 0;
			else if (isinf(output) != 0) output = (output > 0) ? DBL_MAX : DBL_MIN;

			chromo->nodes[currentActiveNode]->output = output;
		}
	}

	/* Set the chromosome outputs */
	for (i = 0; i < numOutputs; i++) {
		if (chromo->outputNodes[i] < numInputs) {
			chromo->outputValues[i] = inputs;
		}
		else {
			chromo->outputValues[i] = chromo->nodes[chromo->outputNodes[i] - numInputs]->output;
		}
	}
}


double calculateFitness(struct chromosome *chromo, struct dataset *data) {
	
	int i, j;
	double error = 0.0;

	for(i = 0; i < data->numSamples; i++) {
		executeChromosome(chromo, data->inputs[i]);
		for(j = 0; j < chromo->numOutputs; j++) {
			error += fabs(chromo->outputValues[j] - data->outputs[i]);
		}
	}
	chromo->fitness = error;
	return error;
}

/* -------------------------------------------------- */


struct dataset *loadDataset(char *fileName) {

	FILE *file;
	int i;
	struct dataset *dset;
	char buffer[100];

	file = fopen(fileName, "r");
	if (!file) {
		printf("Error: file %s was not found.\nExiting.\n", fileName);
		exit(0);
	}

	fgets(buffer, 100, file);
	
	dset = (struct dataset *)malloc(sizeof(struct dataset));

	dset->numInputs  = atoi(strtok(buffer, ","));
	dset->numOutputs = atoi(strtok(NULL, ","));
	dset->numSamples = atoi(strtok(NULL, ","));

	dset->inputs = (double*)malloc(dset->numSamples * sizeof(double));
	dset->outputs = (double*)malloc(dset->numSamples * sizeof(double));

	for(i = 0; i < dset->numSamples; i++) {
		fgets(buffer, 100, file);
		dset->inputs[i] = atof(strtok(buffer, ","));
		dset->outputs[i] = atof(strtok(NULL, ","));
	}

	fclose(file);

	return dset;
}


void freeDataset(struct dataset *data) {
	free(data->inputs);
	free(data->outputs);
	free(data);
}


/* -------------------------------------------------- */


int randint(int min, int max) {

	return rand() % (max-min) + min;
}


float randfloat(float min, float max) {

	return (rand() / (float) RAND_MAX) * (max - min) + min;
}


struct parameters *initialiseParameters(int numNodes, int arity, int numFunctions, struct dataset *data) {
	
	struct parameters *params;

	if (data == NULL) {
		printf("Error: Dataset not initialised.\nExiting.\n");
		exit(0);
	}

	params = (struct parameters*)malloc(sizeof(struct parameters));

	params->numInputs = data->numInputs;
	params->numOutputs = data->numOutputs;
	params->numNodes = numNodes;
	params->arity = arity;
	params->numFunctions = numFunctions;

	return params;
}


void printParameters(struct parameters *params) {
	printf("Inputs: %d\n", params->numInputs);
	printf("Outputs: %d\n", params->numOutputs);
	printf("Nodes: %d\n", params->numNodes);
	printf("Max Arity: %d\n", params->arity);
	printf("Functions: %d\n", params->numFunctions);
}


/* ------------------------- */
/*         CUDA PART         */
/* ------------------------- */


struct absminus {
	__host__ __device__
	double operator()(double &a, double &b) const {
		return fabs(a - b);
	}
};


__host__ void CUDAcreateArrayChromosome(thrust::host_vector<int> &array, struct parameters *params) {
	int i;
	for(i = 0; i < params->numNodes; i++) {
		array[i * (params->arity + 1)] = randint(0, params->numFunctions);
		array[i * (params->arity + 1) + 1] = randint(0, params->numInputs + i);
		array[i * (params->arity + 1) + 2] = randint(0, params->numInputs + i);
	}
	for(i = 0; i < params->numOutputs; i++) {
		array[params->numNodes * (params->arity + 1) + i] = randint(0, params->numInputs + params->numNodes);
	}
}


__global__ void CUDAcalculateChromosomeOutputs(int *solution, double *inputs, double *outputs, int numSamples, int numInputs, int numNodes) {

	int sample = blockIdx.x * blockDim.x + threadIdx.x;

	if (sample < numSamples) {
		int i;
		
		double nodeOutputs[NUMNODES];

		for(i = 0; i < 9; i++) {

			nodeOutputs[i] = 0.0;

			int function = solution[i*3];

			int inIdx1 = solution[i*3+1];
			int inIdx2 = solution[i*3+2];

			double inValue1 = (inIdx1 < numInputs) ? inputs[sample] : nodeOutputs[inIdx1 - numInputs];
			double inValue2 = (inIdx2 < numInputs) ? inputs[sample] : nodeOutputs[inIdx2 - numInputs];

			double output = 0.0;
			switch(function) {
				case ADD:
					output = inValue1 + inValue2;
					break;
				case SUB:
					output = inValue1 - inValue2;
					break;
				case MUL:
					output = inValue1 * inValue2;
					break;
				case DIV:
					output = inValue1 / inValue2;
					break;
			}

			if (isnan(output) != 0) output = 0;
			else if (isinf(output) != 0) output = (output > 0) ? DBL_MAX : DBL_MIN;

			nodeOutputs[i] = output;
		}
		outputs[sample] = nodeOutputs[solution[27]-numInputs];
	}
}


__host__ double CUDAcalculateFitness(thrust::host_vector<int> &h_solution, thrust::device_vector<double> &d_inputs, thrust::device_vector<double> &d_outputs, int numSamples, int numInputs, int numNodes) {

	int *dSolPtr;
	double *dInPtr;
	double *dOutPtr;

	thrust::device_vector<int> d_solution(h_solution.begin(), h_solution.end());
	thrust::device_vector<double> chromoOutputs(numSamples);

	dSolPtr = thrust::raw_pointer_cast(d_solution.data());
	dInPtr  = thrust::raw_pointer_cast(d_inputs.data());
	dOutPtr = thrust::raw_pointer_cast(chromoOutputs.data());

	int numThreads = NTHREADS;
	int numBlocks = ceil((float)numSamples/numThreads);
	CUDAcalculateChromosomeOutputs<<<numBlocks, numThreads>>>(dSolPtr, dInPtr, dOutPtr, numSamples, numInputs, numNodes);

	thrust::transform(d_outputs.begin(), d_outputs.end(), 
		chromoOutputs.begin(), 
		chromoOutputs.begin(),
		absminus());

	return thrust::reduce(chromoOutputs.begin(), chromoOutputs.end());
}


__host__ void CUDAsingleMutation(thrust::host_vector<int> &solution, struct parameters *params) {
	
	int size = params->numNodes * (params->arity + 1) + params->numOutputs;
	int geneToMutate, nodeIndex, subIndex, oldValue, newValue;

	do {
		/* picks a random gene to mutate */
		geneToMutate = randint(0, size);

		nodeIndex = geneToMutate/(params->arity+1);
		subIndex = geneToMutate%(params->arity+1);

		/*store the old value */
		oldValue = solution[geneToMutate];

		/* mutate normal node */
		if (nodeIndex < params->numNodes) {
			/* mutate function gene */
			if (subIndex == 0) {
				newValue = randint(0, params->numFunctions);
			}
			/* mutate input gene */
			else {
				newValue = getRandomNodeInput(params->numInputs, nodeIndex);
			}
		}
		/* mutate output node */
		else {
			newValue = getRandomChromosomeOutput(params->numInputs, params->numNodes);
		}
	} while(oldValue == newValue);

	solution[geneToMutate] = newValue;
}


__host__ int *CUDAexecuteCGP(struct parameters *params, struct dataset *data, int popSize, int numGens) {

	int i, j, solSize; int *result;
	double bestFit, tempFit;

	solSize = params->numNodes * (params->arity + 1) + params->numOutputs;

	/* host arrays that handles solutions */
	thrust::host_vector<int> best(solSize);
	thrust::host_vector<int> temp(solSize);

	/* device arrays for the data inputs and outputs */
	thrust::device_vector<double> d_inputs(data->inputs, data->inputs + data->numSamples);
	thrust::device_vector<double> d_outputs(data->outputs, data->outputs + data->numSamples);

	/* creates popSize chromosomes and stores the best one */
	CUDAcreateArrayChromosome(best, params);

	bestFit = CUDAcalculateFitness(best, d_inputs, d_outputs, data->numSamples, params->numInputs, params->numNodes);

	for(i = 0; i < popSize-1; i++) {

		CUDAcreateArrayChromosome(temp, params);

		tempFit = CUDAcalculateFitness(temp, d_inputs, d_outputs, data->numSamples, params->numInputs, params->numNodes);

		if(tempFit <= bestFit) {
			/* copying temp --> best */
			thrust::copy(temp.begin(), temp.end(), best.begin());
			bestFit = tempFit;
		}
	}

	for(i = 0; i < numGens; i++) {

		for(j = 0; j < popSize-1; j++) {

			/* copies the best to mutate best --> temp */
			thrust::copy(best.begin(), best.end(), temp.begin());

			/* applies the mutation */
			CUDAsingleMutation(temp, params);
			tempFit = CUDAcalculateFitness(temp, d_inputs, d_outputs, data->numSamples, params->numInputs, params->numNodes);
			
			/* if a mutated chromosome is better than best, save it  */
			if(tempFit <= bestFit) {
				/* copying temp --> best */
				thrust::copy(temp.begin(), temp.end(), best.begin());
				bestFit = tempFit;
			}
		}
	}

	result = (int*)malloc(solSize * sizeof(int));
	thrust::copy(best.begin(), best.end(), result);
	return result;
}

#endif