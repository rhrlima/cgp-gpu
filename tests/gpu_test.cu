#include "hip/hip_runtime.h"
#include <time.h>
#include "../cuda/cgp.cuh"

#define DATASETBUFFER 100

#define NUMNODES 9
#define MAXARITY 2
#define NUMFUNCTIONS 4

#define POPSIZE 5
#define MAXGENS 100

#define RUNS 10

int main(int argc, char *argv[]) {

	unsigned int seed = time(NULL);
	srand(seed);

	char dataset_file[DATASETBUFFER];

	struct dataset *data;
	struct parameters *params;

	int *result;

	if (argc > 1) strcpy(dataset_file, argv[1]);
	else exit(0);

	data = loadDataset(dataset_file);
	
	params = initialiseParameters(NUMNODES, MAXARITY, NUMFUNCTIONS, data);

	clock_t start, end;
	double interval;

	int i;
	for(i = 0; i < RUNS; i++) {
		start = clock();
		result = CUDAexecuteCGP(params, data, POPSIZE, MAXGENS);
		end = clock();
		interval = (double) (end - start) / CLOCKS_PER_SEC;
		printf("%f\n", interval);
	}

	free(result);
	freeDataset(data);
	free(params);

	return 0;
}